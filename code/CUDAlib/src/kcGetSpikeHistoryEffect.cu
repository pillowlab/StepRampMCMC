#include "hip/hip_runtime.h"

#include <math.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include <hiprand.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#include "mex.h"

#include "kcDefs.h" //see for info on anything starting with KC_
#include "kcArrayFunctions.h"


// computes spike history effect
__global__ void kcSpikeHistoryEffect(KC_FP_TYPE * y, int * trIdx, KC_FP_TYPE * y_hist, KC_FP_TYPE * h_filt, int NH, int NT, KC_FP_TYPE * spe) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < NT) {

        for(int ii = trIdx[idx]; ii < trIdx[idx+1]; ii++)  {
            
            spe[ii] = 0;

            for(int jj = 0; jj < NH; jj++) {
                
                if(jj < NH && ii<(trIdx[idx]+jj+1)) {
                    spe[ii] += y_hist[NH*(idx+1) + (ii-trIdx[idx]) - jj-1]*h_filt[jj];
                }
                else if(jj < NH) {
                    spe[ii] += y[ii-jj-1]*h_filt[jj];
                }

            }

        }

    }

}

//Computes the spike history effect
//args
//  0  = y (observations, on GPU)
//  1  = trIdx
//  2  = spike history (spikes before start of trials, NH*NT x 1)
//  3  = spike history filters
//
//outputs (left-hand side)
//  0  = spike history effect pointer

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])  {
    hipError_t ce;

    //loads up trial information
    unsigned int TT = kcGetArrayNumEl(prhs[0]);
    unsigned int NT = kcGetArrayNumEl(prhs[1])-1;
    KC_FP_TYPE * y      = kcGetArrayData(prhs[0],TT);
    int * trIdx = kcGetArrayDataInt(prhs[1]);

    int NH  = mxGetNumberOfElements(prhs[3]);

    //loads filter values
    KC_FP_TYPE * h_filt;
    checkCudaErrors(hipMalloc((void**)&h_filt,sizeof(KC_FP_TYPE)*NH));   
    checkCudaErrors(hipMemcpy(h_filt,(KC_FP_TYPE*)mxGetPr(prhs[3]),sizeof(KC_FP_TYPE)*NH,hipMemcpyHostToDevice));

    //loads spike history before trials
    KC_FP_TYPE * y_hist = kcGetArrayData(prhs[2],NH*NT);

    // sets up space for spike history effect
    KC_FP_TYPE * spe;
    checkCudaErrors(hipMalloc((void**)&spe,sizeof(KC_FP_TYPE)*TT));   

    //sets up CUDA variables
    int blockSize = 2;
    int numBlocks = (int)NT/(int)blockSize + ((NT%blockSize==0)?0:1);

    // computes spike history effect
    kcSpikeHistoryEffect<<< numBlocks,blockSize >>>(y,trIdx,y_hist,h_filt,NH,NT,spe);
    checkCudaErrors(hipDeviceSynchronize());

    // push pointer to matlab
    mwSize dims[2] = {TT, 1};
    plhs[0] = kcSetupEmptyArray(2,dims);
    unsigned KC_PTR_SIZE int * ptr = (unsigned KC_PTR_SIZE int*)mxGetPr(mxGetField(plhs[0],0,KC_ARRAY_PTR));
    *ptr = (unsigned KC_PTR_SIZE int)spe;

    //clears up GPU variables
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(h_filt));

    ce = hipDeviceSynchronize();
    if(ce != hipSuccess) {
        mexPrintf("Error at the end of kcLangevinStep.cu ");
        mexPrintf(hipGetErrorString(ce));
        mexPrintf(" (%d)\n", (int)ce);
        mexErrMsgTxt("CUDA errors");
    }

}